#include "hip/hip_runtime.h"
/* 
Code for the equation solver. 
Author: Harshvardhan Agrawal
*/

#include <stdio.h>
#include <string.h>
#include <malloc.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "grid.h" // This file defines the grid data structure
#include <sys/time.h>
// includes, kernels
#include "solver_kernel.cu"

extern "C" void compute_gold(GRID_STRUCT *);

struct timeval start,stop,start_gpu,stop_gpu,start_gpu_texture,stop_gpu_texture;

float gpu_run_time_global,gpu_run_time_texture;

/* This function prints the grid on the screen */
void 
display_grid(GRID_STRUCT *my_grid)
{
	for(int i = 0; i < my_grid->dimension; i++)
		for(int j = 0; j < my_grid->dimension; j++)
			printf("%f \t", my_grid->element[i * my_grid->dimension + j]);
   		
		printf("\n");
}


/* This function prints out statistics for the converged values, including min, max, and average. */
void 
print_statistics(GRID_STRUCT *my_grid)
{
		// Print statistics for the CPU grid
		float min = INFINITY;
		float max = 0.0;
		double sum = 0.0; 
		for(int i = 0; i < my_grid->dimension; i++){
			for(int j = 0; j < my_grid->dimension; j++){
				sum += my_grid->element[i * my_grid->dimension + j]; // Compute the sum
				if(my_grid->element[i * my_grid->dimension + j] > max) max = my_grid->element[i * my_grid->dimension + j]; // Determine max
				if(my_grid->element[i * my_grid->dimension + j] < min) min = my_grid->element[i * my_grid->dimension + j]; // Determine min
				 
			}
		}

	printf("AVG: %f \n", sum/(float)my_grid->num_elements);
	printf("MIN: %f \n", min);
	printf("MAX: %f \n", max);

	printf("\n");
}


/* Calculate the differences between grid elements for the various implementations. */
void compute_grid_differences(GRID_STRUCT *grid_1, GRID_STRUCT *grid_2)
{
    float diff;
    int dimension = grid_1->dimension;
    int num_elements = dimension*dimension;

    diff = 0.0;
    for(int i = 0; i < grid_1->dimension; i++){
        for(int j = 0; j < grid_1->dimension; j++){
            diff += fabsf(grid_1->element[i * dimension + j] - grid_2->element[i * dimension + j]);
        }
    }
    printf("Average difference in grid elements for Gauss Seidel and Jacobi methods = %f. \n", \
            diff/num_elements);

}



/* This function creates a grid of random floating point values bounded by UPPER_BOUND_ON_GRID_VALUE */
void 
create_grids(GRID_STRUCT *grid_for_cpu, GRID_STRUCT *grid_for_gpu)
{
	printf("Creating a grid of dimension %d x %d. \n", grid_for_cpu->dimension, grid_for_cpu->dimension);
	grid_for_cpu->element = (float *)malloc(sizeof(float) * grid_for_cpu->num_elements);
	grid_for_gpu->element = (float *)malloc(sizeof(float) * grid_for_gpu->num_elements);


	srand((unsigned)time(NULL)); // Seed the the random number generator 
	
	float val;
	for(int i = 0; i < grid_for_cpu->dimension; i++)
		for(int j = 0; j < grid_for_cpu->dimension; j++){
			val =  ((float)rand()/(float)RAND_MAX) * UPPER_BOUND_ON_GRID_VALUE; // Obtain a random value
			grid_for_cpu->element[i * grid_for_cpu->dimension + j] = val; 	
			grid_for_gpu->element[i * grid_for_gpu->dimension + j] = val; 				
		}
}

GRID_STRUCT* 
AllocateDeviceGrid(GRID_STRUCT* my_grid)                        /* Allocate a device grid of same size as my_grid. */
{
	GRID_STRUCT* d_grid;
	d_grid = (GRID_STRUCT *)malloc(sizeof(GRID_STRUCT));
	int size = my_grid->num_elements*sizeof(float);
	hipMalloc((void**)&d_grid->element,size);
	printf("Allocated memory for elements\n");
	return d_grid;
}

void 
CopyToDeviceGrid(GRID_STRUCT* d_grid, GRID_STRUCT* my_grid)      /* Copy a host grid to a device grid. */
{
	printf("Entering copy to device grid \n");
	d_grid->dimension = my_grid->dimension;
	d_grid->num_elements = my_grid->num_elements;
	int size = my_grid->num_elements*sizeof(float);
	hipMemcpy(d_grid->element, my_grid->element,size, hipMemcpyHostToDevice);
}

void CopyFromDeviceGrid(GRID_STRUCT* my_grid,GRID_STRUCT* d_grid){
	int size = my_grid->num_elements*sizeof(float);
	hipMemcpy(my_grid->element, d_grid->element, size, hipMemcpyDeviceToHost);
}

void FreeDeviceGrid(GRID_STRUCT* d_grid){
	hipFree(d_grid->element);
	d_grid->element = NULL;
}

void 
checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}

/* Edit this function skeleton to solve the equation on the device. Store the results back in the my_grid->element data structure for comparison with the CPU result. */
void 
compute_on_device(GRID_STRUCT *my_grid)
{
	printf("Entering compute on device using global memory\n");
	GRID_STRUCT* d_grid1 = AllocateDeviceGrid(my_grid);                    /* Load my_grid to the device. */
	CopyToDeviceGrid(d_grid1, my_grid);

	GRID_STRUCT* d_grid2 = AllocateDeviceGrid(my_grid);
	CopyToDeviceGrid(d_grid2, my_grid);

	float *h_diff;
	h_diff = (float *)malloc(sizeof(float));
	*h_diff = GRID_DIMENSION*GRID_DIMENSION;
	float *d_diff;
//	int size_temp = GRID_DIMENSION*GRID_DIMENSION*sizeof(float);
//	h_diff = (float *)malloc(size_temp);
	hipMalloc((void **)&d_diff,sizeof(float));
/*	for (int i=0;i<GRID_DIMENSION*GRID_DIMENSION;i++){
		h_diff[i]=0.0;
	}
	hipMemcpy(d_diff,h_diff,size_temp,hipMemcpyHostToDevice);
*/
	// Allocate space for the lock on the GPU and initialize it
	int *mutex_on_device=NULL;
	hipMalloc((void**)&mutex_on_device,sizeof(int));
	hipMemset(mutex_on_device,0,sizeof(int));

	// Set up the execution grid
	dim3 threads(TILE_SIZE,TILE_SIZE);
	dim3 grid(GRID_DIMENSION/threads.x,GRID_DIMENSION/threads.y);
	int done=0;
	int numiters=0;
	
	printf("Launching kernel loop\n");
	// Launch the kernel
	gettimeofday(&start_gpu,NULL);
	while(!done){
		hipMemset(d_diff,0.0f,sizeof(float));
		numiters++;
		solver_kernel_naive<<< grid, threads>>>(d_grid1->element,d_grid2->element,GRID_DIMENSION,d_diff,mutex_on_device);
		hipDeviceSynchronize();
		checkCUDAError("Error in kernel");   
		hipMemcpy(h_diff,d_diff,sizeof(float),hipMemcpyDeviceToHost);
	/*	for(int k=0;k<GRID_DIMENSION*GRID_DIMENSION;k++){
			diff = diff + h_diff[k]; 	
		}*/
		if (*h_diff/(float)(GRID_DIMENSION*GRID_DIMENSION) < (float)TOLERANCE) done=1;
		printf("numiters:%d done=%d diff=%f\n",numiters,done,*h_diff);
	}
	gettimeofday(&stop_gpu,NULL);
	
	gpu_run_time_global = (float)(stop_gpu.tv_sec - start_gpu.tv_sec + (stop_gpu.tv_usec - start_gpu.tv_usec)/(float)(1000000));
	
	CopyFromDeviceGrid(my_grid,d_grid1);
	
	hipFree(mutex_on_device);
	hipFree(d_diff);
	FreeDeviceGrid(d_grid2);
	FreeDeviceGrid(d_grid1);
	free(h_diff);
	free(d_grid2);
	free(d_grid1);
}

void 
compute_on_device_texture(GRID_STRUCT *my_grid_1)
{
	printf("Entering compute on device using texture memory\n");
	GRID_STRUCT* d_grid1 = AllocateDeviceGrid(my_grid_1);                    /* Load my_grid to the device. */
	CopyToDeviceGrid(d_grid1, my_grid_1);

	GRID_STRUCT* d_grid2 = AllocateDeviceGrid(my_grid_1);
	CopyToDeviceGrid(d_grid2, my_grid_1);

	float *h_diff;
	h_diff = (float *)malloc(sizeof(float));
	*h_diff = GRID_DIMENSION*GRID_DIMENSION;
	
	float *d_diff;
	hipMalloc((void **)&d_diff,sizeof(float));
	

	// Allocate space for the lock on the GPU and initialize it
	int *mutex_on_device=NULL;
	hipMalloc((void**)&mutex_on_device,sizeof(int));
	hipMemset(mutex_on_device,0,sizeof(int));

	//Bind grid1 and grid2 elements to textures
/*	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc desc1 = hipCreateChannelDesc<float>();
	hipBindTexture2D(NULL, grid1_on_tex_2D, d_grid1->element, desc, d_grid1->dimension, d_grid1->dimension, d_grid1->dimension*sizeof(float));
	hipBindTexture2D(NULL, grid2_on_tex_2D, d_grid2->element, desc1, d_grid2->dimension, d_grid2->dimension, d_grid2->dimension*sizeof(float));
*/
	hipBindTexture(NULL, grid1_on_tex, d_grid1->element, d_grid1->num_elements*sizeof(float));
	hipBindTexture(NULL, grid2_on_tex, d_grid2->element, d_grid2->num_elements*sizeof(float));

	// Set up the execution grid
	dim3 threads(TILE_SIZE,TILE_SIZE);
	dim3 grid(GRID_DIMENSION/threads.x,GRID_DIMENSION/threads.y);
	int done=0;
	int numiters=0;

	printf("Launching kernel loop\n");
	// Launch the kernel
	gettimeofday(&start_gpu_texture,NULL);
	while(!done){
		hipMemset(d_diff,0.0f,sizeof(float));
		numiters++;
		solver_kernel_optimized<<< grid, threads>>>(d_grid1->element,d_grid2->element,GRID_DIMENSION,d_diff,mutex_on_device);
		hipDeviceSynchronize();
		checkCUDAError("Error in kernel");   
		hipMemcpy(h_diff,d_diff,sizeof(float),hipMemcpyDeviceToHost);
	/*	for(int k=0;k<GRID_DIMENSION*GRID_DIMENSION;k++){
			diff = diff + h_diff[k]; 	
		}*/
		if (*h_diff/(float)(GRID_DIMENSION*GRID_DIMENSION) < (float)TOLERANCE) done=1;
		printf("numiters:%d done=%d diff=%f\n",numiters,done,*h_diff);
	}
	gettimeofday(&stop_gpu_texture,NULL);
	
	gpu_run_time_texture = (float)(stop_gpu_texture.tv_sec - start_gpu_texture.tv_sec + (stop_gpu_texture.tv_usec - start_gpu_texture.tv_usec)/(float)(1000000));
	
	CopyFromDeviceGrid(my_grid_1,d_grid1);
	
	
	hipFree(d_diff);
	hipUnbindTexture(grid1_on_tex);
	hipUnbindTexture(grid2_on_tex);
	FreeDeviceGrid(d_grid2);
	FreeDeviceGrid(d_grid1);
	free(h_diff);
	free(d_grid2);
	free(d_grid1);
}

/* The main function */
int 
main(int argc, char **argv)
{	
	/* Generate the grid */
	GRID_STRUCT *grid_for_cpu = (GRID_STRUCT *)malloc(sizeof(GRID_STRUCT)); // The grid data structure
	GRID_STRUCT *grid_for_gpu = (GRID_STRUCT *)malloc(sizeof(GRID_STRUCT)); // The grid data structure
	GRID_STRUCT *grid_for_gpu_texture = (GRID_STRUCT *)malloc(sizeof(GRID_STRUCT));

	grid_for_cpu->dimension = GRID_DIMENSION;
	grid_for_cpu->num_elements = grid_for_cpu->dimension * grid_for_cpu->dimension;
	grid_for_gpu->dimension = GRID_DIMENSION;
	grid_for_gpu->num_elements = grid_for_gpu->dimension * grid_for_gpu->dimension;
	grid_for_gpu_texture->dimension = GRID_DIMENSION;
	grid_for_gpu_texture->num_elements = grid_for_gpu_texture->dimension * grid_for_gpu_texture->dimension;

	grid_for_gpu_texture->element = (float *)malloc(sizeof(float)*grid_for_gpu_texture->num_elements);
 	create_grids(grid_for_cpu, grid_for_gpu); // Create the grids and populate them with the same set of random values
	printf("Creating grid for texture memory\n");	
	
	for (int i=0;i<GRID_DIMENSION;i++) {
		for(int j=0;j<GRID_DIMENSION;j++){
			grid_for_gpu_texture->element[i*GRID_DIMENSION + j] = grid_for_gpu->element[i*GRID_DIMENSION + j];
		}
	}
	printf("Using the cpu to solve the grid. \n");
	gettimeofday(&start,NULL);
	compute_gold(grid_for_cpu);  // Use CPU to solve 
	gettimeofday(&stop,NULL);
	float cpu_run_time = (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)(1000000));
	
	// Use the GPU to solve the equation
	compute_on_device(grid_for_gpu);
	
	compute_on_device_texture(grid_for_gpu_texture);	
	
	// Print key statistics for the converged values
	printf("CPU: \n");
	print_statistics(grid_for_cpu);

	printf("GPU: \n");
	print_statistics(grid_for_gpu);

	printf("GPU with texture: \n");
	print_statistics(grid_for_gpu_texture);	
    /* Compute grid differences. */
    	compute_grid_differences(grid_for_cpu, grid_for_gpu);
	compute_grid_differences(grid_for_cpu,grid_for_gpu_texture);
	printf("CPU run time: %0.4fs\n",cpu_run_time);
	printf("GPU run time with global memory: %0.4fs\n",gpu_run_time_global);
	printf("Speedup (global memory): %0.4f\n",(cpu_run_time/gpu_run_time_global));

	printf("GPU run time with texture memory: %0.4fs\n",gpu_run_time_texture);
	printf("Speedup (texture memory): %0.4f\n",(cpu_run_time/gpu_run_time_texture));	
	
	free((void *)grid_for_cpu->element);	
	free((void *)grid_for_cpu); // Free the grid data structure 
	
	free((void *)grid_for_gpu->element);	
	free((void *)grid_for_gpu); // Free the grid data structure 

	free((void *)grid_for_gpu_texture->element);
	free((void *)grid_for_gpu_texture);
	exit(0);
}
